﻿#include <stdio.h>
#include <iostream>
#include <vector>
#include <random>
#include <iomanip>

#include "hip/hip_runtime.h"


using namespace std;

template <typename T>
void printVector(const T *a, int size)
{
    for (int i = 0; i < size; i++)
        cout << setw(3) << a[i];
    cout << endl;
}

__global__ void addKernel(const int *a, const int *b, int *c, int size)
{
    // int i = threadIdx.x;
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < size)
        c[i] = a[i] + b[i];

    // printf("ThreadIdx(% u, % u, % u)\n", threadIdx.x, threadIdx.y, threadIdx.z);
}

int main()
{
    const int threadsPerBlock = 256; // 최대 deviceProp.maxThreadsPerBlock = 1024 까지 가능

    const int size = 1024 * 1024 * 256; // 여기서는 블럭을 여러 개 사용해야 하는 큰 size
    // const int size = 37;

    // 생각해볼 점: 블럭이 몇 개가 필요할까?

    // vector<int> a(size);
    // vector<int> b(size);
    // vector<int> c_single(size);     // 결과 확인용
    // vector<int> c(size, -1); // CUDA에서 계산한 결과 저장

    int *a = nullptr;
    int *b = nullptr;
    int *c_single = nullptr;
    int *c = nullptr;

    // cudaMallocHost 사용해서 gpu 아니라, cpu 메모리를 동작으로 할당
    // 그런데, cpu 메모리인데, pinned-memory를 할당 받음
    // OS가 안쓰는 cpu 메모리는 하드 등의 저장장치로 옮겨두는데,
    // pinned-memory 는 못옮기게 고정시켜서, 항상 1차 저장장치, 메모리에 위치함
    // 그래서, pinned-memory로 잡을수 있는 최대 크기는,
    // 물리적으로 RAM 크기를 넘을수 없고, RAM 크기에서 OS 사용하는 메모리 크기를 뺀.. 크기임
    // 그래서, **GPU로 보낼 데이터를 저장해두는 변수는 보통, cpu의 pinned-memory 에 할당하는 것을 권장**
    hipHostMalloc(&a, sizeof(int) * size, hipHostMallocDefault); // pinned-memory
    hipHostMalloc(&b, sizeof(int) * size, hipHostMallocDefault);
    hipHostMalloc(&c_single, sizeof(int) * size, hipHostMallocDefault); // GPU 통신에 사용되지 않기 때문에 꼭 pinned-memory를 사용할 필요는 없음
    hipHostMalloc(&c, sizeof(int) * size, hipHostMallocDefault);

    for (int i = 0; i < size; i++)
    {
        a[i] = rand() % 10;
        b[i] = rand() % 10;
        c_single[i] = a[i] + b[i];
    }

    cout << "Add vectors using CUDA" << endl;

    {
        // 참고: cudaStreamSynchronize()를 사용해서 개별 스트림만 따로 동기화 하고 싶은 경우
        //      여기서는 cudaDeviceSynchronize()를 사용하기 때문에 스트림만 따로 동기화하지는 않았습니다.
        //{
        //	cudaSetDeviceFlags(cudaDeviceScheduleBlockingSync);

        //	unsigned int flags; cudaError_t err = cudaGetDeviceFlags(&flags);
        //	if (err != cudaSuccess) {
        //		cerr << "Failed to get device flags: " << cudaGetErrorString(err) << endl; return 1;
        //	}
        //	if (flags & cudaDeviceScheduleBlockingSync) {
        //		cout << "cudaDeviceScheduleBlockingSync: set" << endl;
        //	}
        //	else {
        //		cout << "cudaDeviceScheduleBlockingSync: NOT set" << endl;
        //	}
        //}

        // 스트림은 cudaStream_t 자료형으로 선언하고, cudaStreamCreate() 로 만듦
        // 스트림은 메모리카피(cudaMemcpyAsync), 커널 실행 등에 사용됨
        hipStream_t stream;
        hipStreamCreate(&stream);

        int *dev_a = nullptr;
        int *dev_b = nullptr;
        int *dev_c = nullptr;

        hipMalloc((void **)&dev_a, size * sizeof(int)); // input a
        hipMalloc((void **)&dev_b, size * sizeof(int)); // input b
        hipMalloc((void **)&dev_c, size * sizeof(int)); // output c

        hipEvent_t start, stop; // 시간 측정을 위한 CUDA 이벤트 생성 (시간측정도 Nsight로 할 수 있습니다.)
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start, 0); // 시작 시간 기록 (H2D -> Kernel -> D2H)

        // TODO 완성해야 실행됩니다. 간단합니다.

        // 주의: 뒤에 Async가 붙은 cudaMemcpyAsync() 사용
        // cudaMemcpyAsync(dev_a, a, size * sizeof(int), cudaMemcpyHostToDevice, TODO ); // 비동기적으로 복사 복사
        // cudaMemcpyAsync(dev_b, b, size * sizeof(int), cudaMemcpyHostToDevice, TODO );

        // 이 stream을 이용해서, cpu에서 gpu로 메모리를 복사해라!
        // 비동기 함수이기때문에, gpu가 끝날때까지 cpu는 기다리지 않음
        hipMemcpyAsync(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice, stream); // 비동기적으로 복사 복사
        hipMemcpyAsync(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice, stream);

        int blocks = int(ceil(float(size) / threadsPerBlock)); // 블럭 여러 개 사용
        // addKernel << <blocks, threadsPerBlock, 0, TODO >> > (dev_a, dev_b, dev_c, size);

        // 커널을 실행시킬때도, stream을 사용해서 비동기적으로 실행시킬 수 있음
        //         블럭수, 쓰래드개수, 쉐어드메모리크기, 어떤스트림(흐름)으로실행시킬지
        // 비동기식이기때문에, cpu가 gpu한테 명령만 내리고, gpu가 안끝나도, 자기일 할 수 있음
        addKernel<<<blocks, threadsPerBlock, 0, stream>>>(dev_a, dev_b, dev_c, size);

        // 안내:
        // - 커널 호출할때 stream을 지정해주지 않으면 내부적으로 기본 스트림을 사용합니다.
        // - cudaMemcpy()와 달리 커널 호출은 항상 비동기적입니다. GPU에게 명령만 내리고 CPU는 바로 다음 명령을 수행합니다.
        // - CPU에게 GPU가 일을 다 끝날때까지 강제로 기다리게 하고 싶다면 아래의 cudaDeviceSynchronize()를 사용할 수 있습니다.
        // - 함수 이름에서 볼 수 있듯이, 이렇게 기다리는 것을 "동기화(synchronize)"라고 합니다.

        // 결과 복사 device -> host
        // cudaMemcpyAsync(c, dev_c, size * sizeof(int), cudaMemcpyDeviceToHost, TODO );
        // 스트림을 이용해서 계산 결과를 받아옴
        hipMemcpyAsync(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost, stream);

        // **스트림 사용하는 이유**
        // 1) GPU는 GPU대로 따로 일하고, CPU는 그 사이에 다른 일을 할 수 있음
        //      다른 성질의 디바이스, CPU와 GPU를 패러럴하게 비동기적으로 사용한다.
        // 2) 데이터 통신을 하는 동안, 커널을 실행 시킬 수 있음
        //      전체적으로 실행시간을 줄일 수 있음
        // 그런데, 스트림 1개는 직렬(serial)로 동작하므로,
        // 제대로 병렬로 동작시키려면, 여러 스트림을 사용해야 함

        hipEventRecord(stop, 0); // 끝나는 시간 기록
        hipDeviceSynchronize();  // kernel이 끝날때까지 대기 (동기화)

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop); // 걸린 시간 계산
        cout << "Time elapsed: " << milliseconds << " ms" << endl;

        // 안내: kernel 실행 후 cudaGetLastError() 생략

        // 결과 확인
        if (size < 40)
        { // size가 작을 경우에는 출력해서 확인
            printVector(a, size);
            printVector(b, size);
            printVector(c_single, size);
            printVector(c, size);
        }

        for (int i = 0; i < size; i++)
            if (c_single[i] != c[i])
            {
                cout << "Wrong result" << endl;
                return 1;
            }

        cout << "Correct" << endl;

        hipEventDestroy(start);
        hipEventDestroy(stop);

        hipFree(dev_c);
        hipFree(dev_a);
        hipFree(dev_b);

        hipHostFree(a);
        hipHostFree(b);
        hipHostFree(c_single);
        hipHostFree(c);

        hipDeviceReset();
    }

    return 0;
}
