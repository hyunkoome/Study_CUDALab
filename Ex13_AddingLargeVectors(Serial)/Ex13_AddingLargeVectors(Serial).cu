﻿#include <stdio.h>
#include <iostream>
#include <vector>
#include <random>
#include <iomanip>

#include "hip/hip_runtime.h"


#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d in %s: %s\n", \
                    __FILE__, __LINE__, __func__, hipGetErrorString(err)); \
            exit(1); \
        } \
    } while (0)

using namespace std;

template<typename T>
void printVector(const T* a, int size)
{
	for (int i = 0; i < size; i++)
		cout << setw(3) << a[i];
	cout << endl;
}

__global__ void addKernel(const int* a, const int* b, int* c, int size)
{
	// int i = threadIdx.x;
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < size)
		c[i] = a[i] + b[i];

	// printf("ThreadIdx(% u, % u, % u)\n", threadIdx.x, threadIdx.y, threadIdx.z);
}

int main()
{
	const int threadsPerBlock = 1024; // 최대 deviceProp.maxThreadsPerBlock = 1024 까지 가능

	const int size = 1024 * 1024 * 512; // size가 너무 커서 한 번에 모두 계산할 수 없다고 가정
	//const int size = 40;
	const int numSplits = 8;            // 여러 조각으로 나눠서 계산 
	const int split_size = size / numSplits;
	
	int* a = nullptr;
	int* b = nullptr;
	int* c = nullptr;

	hipHostMalloc(&a, sizeof(int) * size, hipHostMallocDefault); // pinned-memory
	hipHostMalloc(&b, sizeof(int) * size, hipHostMallocDefault);
	hipHostMalloc(&c, sizeof(int) * size, hipHostMallocDefault);

	for (int i = 0; i < size; i++)
	{
		a[i] = rand() % 10;
		b[i] = rand() % 10;
	}

	cout << "Add vectors using CUDA" << endl;

	{
		hipStream_t stream;
		hipStreamCreate(&stream);

		int* dev_a = nullptr;
		int* dev_b = nullptr;
		int* dev_c = nullptr;

		hipMalloc((void**)&dev_a, split_size * sizeof(int)); // size -> split_size (GPU 메모리 적게 사용)
		hipMalloc((void**)&dev_b, split_size * sizeof(int)); // size -> split_size (GPU 메모리 적게 사용)
		hipMalloc((void**)&dev_c, split_size * sizeof(int)); // size -> split_size (GPU 메모리 적게 사용)
		 
		hipEvent_t start, stop;// 시간 측정을 위한 CUDA 이벤트 생성 (시간측정도 Nsight로 할 수 있습니다.)
		hipEventCreate(&start);
		hipEventCreate(&stop);

		hipEventRecord(start, 0); // 시작 시간 기록 (H2D -> Kernel -> D2H)

		for (int s = 0; s < numSplits; s++) 
		{
			//cudaMemcpyAsync(dev_a, &a[s * TODO ], TODO * sizeof(int), cudaMemcpyHostToDevice, stream); // size -> split_size
			//cudaMemcpyAsync(dev_b, &b[s * TODO ], TODO * sizeof(int), cudaMemcpyHostToDevice, stream); // size -> split_size

			//int blocks = int(ceil(float( TODO ) / threadsPerBlock)); // 블럭 여러 개 사용
			//addKernel << <blocks, threadsPerBlock, 0, stream >> > (dev_a, dev_b, dev_c, TODO );
			
			//cudaMemcpyAsync(&c[s * TODO ], dev_c, TODO * sizeof(int), cudaMemcpyDeviceToHost, stream);
		}

		hipEventRecord(stop, 0);  // 끝나는 시간 기록
		hipDeviceSynchronize();   // kernel이 끝날때까지 대기 (동기화)

		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop); // 걸린 시간 계산
		cout << "Time elapsed: " << milliseconds << " ms" << endl; // 600ms

		// 안내: kernel 실행 후 cudaGetLastError() 생략

		// 결과 확인
		if (size <= 40) { // size가 작을 경우에는 출력해서 확인
			printVector(a, size);
			printVector(b, size);
			printVector(c, size);
		}

		for (int i = 0; i < size; i++)
			if (c[i] != a[i] + b[i])
			{
				cout << "Wrong result" << endl;
				return 1;
			}

		cout << "Correct" << endl;

		hipEventDestroy(start);
		hipEventDestroy(stop);

		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);

		hipHostFree(a);
		hipHostFree(b);
		hipHostFree(c);

		hipDeviceReset();
	}


	return 0;
}

