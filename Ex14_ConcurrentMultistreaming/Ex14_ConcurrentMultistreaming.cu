﻿#include <stdio.h>
#include <iostream>
#include <vector>
#include <random>
#include <iomanip>

#include "hip/hip_runtime.h"


#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d in %s: %s\n", \
                    __FILE__, __LINE__, __func__, hipGetErrorString(err)); \
            exit(1); \
        } \
    } while (0)

using namespace std;

template<typename T>
void printVector(const T* a, int size)
{
	for (int i = 0; i < size; i++)
		cout << setw(3) << a[i];
	cout << endl;
}

__global__ void addKernel(const int* a, const int* b, int* c, int size)
{
	// int i = threadIdx.x;
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if (i < size)
		c[i] = a[i] + b[i];

	// printf("ThreadIdx(% u, % u, % u)\n", threadIdx.x, threadIdx.y, threadIdx.z);
}

int main()
{
	const int size = 1024 * 1024 * 512; // size가 너무 커서 한 번에 모두 계산할 수 없다고 가정
	//const int size = 40;
	const int numSplits = 8;           // 여러 조각으로 나눠서 계산 
	const int splitSize = size / numSplits;

	int* a = nullptr;
	int* b = nullptr;
	int* c = nullptr;

	hipHostMalloc(&a, sizeof(int) * size, hipHostMallocDefault); // pinned-memory
	hipHostMalloc(&b, sizeof(int) * size, hipHostMallocDefault);
	hipHostMalloc(&c, sizeof(int) * size, hipHostMallocDefault);

	// CPU에서 할 일도 아래에서 분할 수행할 수 있습니다.
	for (int i = 0; i < size; i++)
	{
		a[i] = rand() % 10;
		b[i] = rand() % 10;
	}

	cout << "Add vectors using CUDA" << endl;

	{
		vector<hipStream_t> streams(numSplits);
		for (int s = 0; s < streams.size(); s++)
			hipStreamCreate(&streams[s]);

		vector<int*> dev_a(numSplits);
		vector<int*> dev_b(numSplits);
		vector<int*> dev_c(numSplits);

		for (int s = 0; s < numSplits; s++) { // GPU 메모리가 넉넉하다고 가정
			hipMalloc((void**)&dev_a[s], splitSize * sizeof(int)); // size -> split_size
			hipMalloc((void**)&dev_b[s], splitSize * sizeof(int)); // size -> split_size
			hipMalloc((void**)&dev_c[s], splitSize * sizeof(int)); // size -> split_size
		}

		hipEvent_t start, stop;// 시간 측정을 위한 CUDA 이벤트 생성 (시간측정도 Nsight로 할 수 있습니다.)
		hipEventCreate(&start);
		hipEventCreate(&stop);

		hipEventRecord(start, 0); // 시작 시간 기록 (H2D -> Kernel -> D2H)

		for (int s = 0; s < numSplits; s++)
		{
			// cudaStreamSynchronize(streams[s]); // 참고용이며 여기서는 사용하지 않습니다.

			//cudaMemcpyAsync(dev_a[s], &a[s * splitSize], splitSize * sizeof(int), cudaMemcpyHostToDevice, TODO ); // size -> split_size
			//cudaMemcpyAsync(dev_b[s], &b[s * splitSize], splitSize * sizeof(int), cudaMemcpyHostToDevice, TODO ); // size -> split_size
		}

		for (int s = 0; s < numSplits; s++)
		{
            int threadsPerBlock = 1024; // 최대 deviceProp.maxThreadsPerBlock = 1024 까지 가능
			int blocks = int(ceil(float(splitSize) / threadsPerBlock)); // 블럭 여러 개 사용
			//addKernel << <blocks, threadsPerBlock, 0, TODO >> > (dev_a[s], dev_b[s], dev_c[s], splitSize);
		}

		for (int s = 0; s < numSplits; s++)
		{
			//cudaMemcpyAsync(&c[s * splitSize], dev_c[s], splitSize * sizeof(int), cudaMemcpyDeviceToHost, TODO );
		}

		//for(int s = 0; s < num_splits; s++)  // 참고용이며 여기서는 사용하지 않습니다.
		//	cudaStreamSynchronize(streams[s]); // 밑에서 cudaDeviceSynchronize() 사용

		hipEventRecord(stop, 0);  // 끝나는 시간 기록
		hipDeviceSynchronize();   // kernel이 끝날때까지 대기 (동기화)

		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop); // 걸린 시간 계산
		cout << "Time elapsed: " << milliseconds << " ms" << endl; // 453ms

		// 안내: kernel 실행 후 cudaGetLastError() 생략

		// 결과 확인
		if (size <= 40) { // size가 작을 경우에는 출력해서 확인
			printVector(a, size);
			printVector(b, size);
			printVector(c, size);
		}

		for (int i = 0; i < size; i++)
			if (c[i] != a[i] + b[i])
			{
				cout << "Wrong result" << endl;
				return 1;
			}

		cout << "Correct" << endl;

		hipEventDestroy(start);
		hipEventDestroy(stop);

		for (int s = 0; s < numSplits; s++)
		{
			hipFree(dev_c[s]);
			hipFree(dev_a[s]);
			hipFree(dev_b[s]);
		}

		hipHostFree(a);
		hipHostFree(b);
		hipHostFree(c);

		hipDeviceReset();
	}

	return 0;
}

